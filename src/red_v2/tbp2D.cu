#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>

#include "tbp2D.h"

#include "redutil2.h"
#include "constants.h"

using namespace std;
using namespace redutil2;


tbp2D::tbp2D(string& path_si, string& path_sd, uint16_t n_ppo, comp_dev_t comp_dev) :
	ode(2, 1, 4, n_ppo, comp_dev)
{
	name = "Singular 2D two-body problem";

	initialize();
	allocate_storage();

    load_solution_info(path_si);
    load_solution_data(path_sd);

    calc_integral();
    tout = t;
}

tbp2D::~tbp2D()
{
	deallocate_storage();
}

void tbp2D::initialize()
{
	h_md       = NULL;
	d_md       = NULL;
	md         = NULL;
}

void tbp2D::allocate_storage()
{
	allocate_host_storage();
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		allocate_device_storage();
	}
}

void tbp2D::allocate_host_storage()
{
	ALLOCATE_HOST_VECTOR((void**)&(h_md), n_obj * sizeof(tbp_t::metadata_t));
}

void tbp2D::allocate_device_storage()
{
	ALLOCATE_DEVICE_VECTOR((void**)&(d_md), n_obj * sizeof(tbp_t::metadata_t));
}

void tbp2D::deallocate_storage()
{
	//NOTE : First always release the DEVICE memory
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		deallocate_device_storage();
	}
	deallocate_host_storage();
}

void tbp2D::deallocate_host_storage()
{
	FREE_HOST_VECTOR((void **)&(h_md));
}

void tbp2D::deallocate_device_storage()
{
	FREE_DEVICE_VECTOR((void **)&(d_md));
}

void tbp2D::copy_metadata(copy_direction_t dir)
{
	switch (dir)
	{
	case COPY_DIRECTION_TO_DEVICE:
		copy_vector_to_device(d_md, h_md, n_obj*sizeof(tbp_t::metadata_t));
		break;
	case COPY_DIRECTION_TO_HOST:
		copy_vector_to_host(h_md, d_md, n_obj*sizeof(tbp_t::metadata_t));
		break;
	default:
		throw std::string("Parameter 'dir' is out of range.");
	}
}

void tbp2D::calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* acc, var_t* jrk)
{
	throw string("The tbp2D::calc_dy is not implemented.");
}

void tbp2D::calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	if (PROC_UNIT_CPU == comp_dev.proc_unit)
	{
		cpu_calc_dy(stage, curr_t, y_temp, dy);
	}
	else
	{
		gpu_calc_dy(stage, curr_t, y_temp, dy);
	}
}

void tbp2D::calc_integral()
{
	static bool first_call = true;
	const tbp_t::param_t* p = (tbp_t::param_t*)h_p;

	var_t r  = sqrt(SQR(h_y[0]) + SQR(h_y[1]));
	var_t v2 = SQR(h_y[2]) + SQR(h_y[3]);
	integral.h = 0.5 * v2 - p[0].mu / r;
	if (first_call)
	{
		integral.h0 = integral.h;
		first_call = false;
	}
}

void tbp2D::cpu_calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	static const var_t mu = h_p[0];

	var_t r = sqrt(SQR(y_temp[0]) + SQR(y_temp[1]));
	var_t r3 = r*r*r;

	dy[0] = y_temp[2];                 // dx1 / dt = x3
	dy[1] = y_temp[3];                 // dx2 / dt = x4

	dy[2] = -(mu / r3) * y_temp[0];    // dx3 / dt = -mu / (r^3) * x1
	dy[3] = -(mu / r3) * y_temp[1];    // dx4 / dt = -mu / (r^3) * x2
}

void tbp2D::gpu_calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	throw string("The gpu_calc_dy() is not implemented.");
}

void tbp2D::load_solution_info(string& path)
{
	ifstream input;

	cout << "Loading " << path << " ";

	data_rep_t repres = file::get_data_repres(path);
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		input.open(path.c_str(), ios::in);
		if (input) 
		{
			input >> t >> dt;
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		input.open(path.c_str(), ios::in | ios::binary);
		if (input) 
		{
    		input.read((char*)&t, sizeof(var_t));
	        input.read((char*)&dt, sizeof(var_t));
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}
	input.close();

	cout << " done" << endl;
}

void tbp2D::load_solution_data(string& path)
{
	ifstream input;

	cout << "Loading " << path << " ";

	data_rep_t repres = file::get_data_repres(path);
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		input.open(path.c_str(), ios::in);
		if (input) 
		{
			load_ascii(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		input.open(path.c_str(), ios::in | ios::binary);
		if (input) 
		{
			load_binary(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}
	input.close();

	cout << " done" << endl;
}

void tbp2D::load_ascii(ifstream& input)
{
	// id
	input >> h_md[0].id;
	// mu = k^2*(m1 + m2)
	input >> h_p[0];
	// x - position
	input >> h_y[0];
	// y - position
	input >> h_y[1];
	// x - velocity
	input >> h_y[2];
	// y - velocity
	input >> h_y[3];
}

void tbp2D::load_binary(ifstream& input)
{
	throw string("The load_binary() is not implemented.");
}

void tbp2D::print_solution(std::string& path_si, std::string& path_sd, data_rep_t repres)
{
	ofstream sout;

	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		sout.open(path_si.c_str(), ios::out | ios::app);
		break;
	case DATA_REPRESENTATION_BINARY:
		sout.open(path_si.c_str(), ios::out | ios::app | ios::binary);
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}
	if (!sout)
	{
		throw string("Cannot open " + path_si + ".");
	}
	file::tbp::print_solution_info(sout, t, dt, repres);
	sout.close();

	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		sout.open(path_sd.c_str(), ios::out | ios::app);
		break;
	case DATA_REPRESENTATION_BINARY:
		sout.open(path_sd.c_str(), ios::out | ios::app | ios::binary);
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}
	if (!sout)
	{
		throw string("Cannot open " + path_sd + ".");
	}
	file::tbp::print_solution_data(sout, n_obj, n_ppo, n_vpo, h_md, h_p, h_y, repres);
	sout.close();
}

void tbp2D::print_dump(std::string& path_si, std::string& path_sd)
{
    throw string("The tbp2D::print_dump function is not implemented.");
}

void tbp2D::print_integral(string& path)
{
	ofstream sout;

	sout.open(path.c_str(), ios::out | ios::app);
	if (sout)
	{
		sout.precision(16);
		sout.setf(ios::right);
		sout.setf(ios::scientific);

	    sout << setw(VAR_T_W) << t << SEP             /* time of the record [day] (double)           */
		     << setw(VAR_T_W) << integral.h << endl;  /* energy of the system                        */
	}
	else
	{
		throw string("Cannot open " + path + ".");
	}
	sout.close();
}
