#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ode.h"
#include "int_rungekutta5.h"

#include "macro.h"
#include "redutil2.h"

using namespace redutil2;

#if 0
/*
 * Fehlberg, E.
 * "Classical Fifth-, Sixth-, Seventh-, and Eighth-Order Runge-Kutta Formulas with Stepsize Control"
 * NASA-TR-R-287 (https://nix.nasa.gov/search.jsp?R=19680027281&qs=N%253D4294957355)
 * p. 26 Table II. RK5(6)
 */
// The Runge-Kutta matrix
var_t int_rungekutta5::a[] = 
{        /*    0            1           2           3             4          5     6  */
/* 0 */         0.0,          0.0,         0.0,         0.0,          0.0,  0.0,  0.0, 
/* 1 */     1.0/6.0,          0.0,         0.0,         0.0,          0.0,  0.0,  0.0, 
/* 2 */     4.0/75.0,   16.0/75.0,         0.0,         0.0,          0.0,  0.0,  0.0, 
/* 3 */     5.0/6.0,    -8.0/3.0,    5.0/2.0,           0.0,          0.0,  0.0,  0.0,
/* 4 */    -8.0/5.0,   144.0/25.0,      -4.0,    16.0/25.0,          0.0,   0.0,  0.0, 
/* 5 */   361.0/320.0,  -18.0/5.0, 407.0/128.0, -11.0/80.0,   55.0/128.0,   0.0,  0.0, 
/* 6 */   -11.0/640.0,        0.0,  11.0/256.0, -11.0/160.0,  11.0/256.0,   0.0,  0.0,
/* 7 */    93.0/640.0,  -18.0/5.0, 803.0/256.0, -11.0/160.0,  99.0/256.0,   0.0,  1.0
};
// weights
var_t int_rungekutta5::bh[] = { 7.0/1408, 0.0, 1125.0/2816.0, 9.0/32.0, 125.0/768.0, 0.0, 5.0/66.0, 5.0/66.0 };
// nodes
var_t int_rungekutta5::c[]  = { 0.0, 1.0/6.0, 4.0/15.0, 2.0/3.0, 4.0/5.0, 1.0, 0.0, 1.0 };
#endif

/*
 * Dormand, J. R.; Prince, P. J.
 * "New Runge-Kutta algorithms for numerical simulation in dynamical astronomy"
 * Celestial Mechanics, vol. 18, Oct. 1978, p. 223-232.
 * p. 225 Table II. Runge-Kutta 5(4)T
 */
static const var_t lambda = 1.0/60.0;

// The Runge-Kutta matrix
var_t int_rungekutta5::a[] = 
{      /*        1              2                 3                 4              5       6      */
/* 1 */         0.0,           0.0,              0.0,              0.0,           0.0,    0.0,  // -> k1
/* 2 */   1.0/8.0,             0.0,              0.0,              0.0,           0.0,    0.0, 	// -> k2
/* 3 */         0.0,     1.0/4.0,                0.0,              0.0,           0.0,    0.0, 	// -> k3
/* 4 */ 196.0/729.0,  -320.0/729.0,    448.0/729.0,                0.0,           0.0,    0.0,	// -> k4
/* 5 */ 836.0/2875.0,   64.0/575.0, -13376.0/20125.0,  21384.0/20125.0,           0.0,    0.0, 	// -> k5
/* 6 */ -73.0/48.0,            0.0,   1312.0/231.0,    -2025.0/448.0,   2875.0/2112.0,    0.0, 	// -> k6
/*-------------------------------------------------------------------------------------------------------*/
/* 7 */  17.0/192.0,           0.0,     64.0/231.0,     2187.0/8960.0,  2875.0/8448.0, 1.0/20	// -> k7
}; /* 7 x 6 matrix */
static uint16_t a_row = 7;
static uint16_t a_col = 6;
// weights
var_t int_rungekutta5::bh[] = { 17.0/192.0, 0.0, 64.0/231.0, 2187.0/8960.0, 2875.0/8448.0, 1.0/20 };
// nodes
var_t int_rungekutta5::c[]  = { 0.0, 1.0/8.0, 1.0/4.0, 4.0/9.0, 4.0/5.0, 1.0, 1.0 };

// These arrays will contain the stepsize multiplied by the constants
var_t int_rungekutta5::h_a[ sizeof(int_rungekutta5::a ) / sizeof(var_t)];
var_t int_rungekutta5::h_bh[ sizeof(int_rungekutta5::bh ) / sizeof(var_t)];

//__constant__ var_t dc_a[sizeof(int_rungekutta5::a) / sizeof(var_t)];
//__constant__ var_t dc_bh[sizeof(int_rungekutta5::bh) / sizeof(var_t)];


int_rungekutta5::int_rungekutta5(ode& f, bool adaptive, var_t tolerance, comp_dev_t comp_dev) :
	integrator(f, adaptive, tolerance, (adaptive ? 7 : 6), comp_dev)
{
	name    = "Runge-Kutta5";
	n_order = 5;

	d_a  = NULL;
	d_bh = NULL;
	check_Butcher_tableau();
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		allocate_Butcher_tableau();
	}	
}

int_rungekutta5::~int_rungekutta5()
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		deallocate_Butcher_tableau();
	}	
}

void int_rungekutta5::allocate_Butcher_tableau()
{
	ALLOCATE_DEVICE_VECTOR((void**)&d_a,  sizeof(a));
	ALLOCATE_DEVICE_VECTOR((void**)&d_bh, sizeof(bh));
}

void int_rungekutta5::deallocate_Butcher_tableau()
{
	FREE_DEVICE_VECTOR((void**)&d_a);
	FREE_DEVICE_VECTOR((void**)&d_bh);
}

void int_rungekutta5::check_Butcher_tableau()
{
	uint16_t n_c = sizeof(int_rungekutta5::c) / sizeof(var_t);
	uint16_t n_col = (sizeof(int_rungekutta5::a) / sizeof(var_t)) / n_c;

	for (uint16_t i = 0; i < n_c; i++)
	{
		var_t sum = 0.0;
		for (uint16_t j = 0; j < n_col; j++)
		{
			uint16_t k = i * n_col + j;
			sum += a[k];
		}
		if (1.0e-15 < fabs(sum - c[i]))
		{
			throw std::string("The Runge-Kutta 5 is not consistent (sum(a_ij) != c_i).");
		}
	}
}

void int_rungekutta5::calc_ytemp(uint16_t stage)
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		var_t* coeff = d_a + stage * a_col;
		gpu_calc_lin_comb_s(ytemp, f.y, d_k, coeff, stage, f.n_var, comp_dev.id_dev, optimize);
	}
	else
	{
		var_t* coeff = h_a + stage * a_col;
		tools::calc_lin_comb_s(ytemp, f.y, h_k, coeff, stage, f.n_var);
	}
}

void int_rungekutta5::calc_y_np1()
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
		var_t* coeff = d_bh;
		gpu_calc_lin_comb_s(f.yout, f.y, d_k, coeff, 6, f.n_var, comp_dev.id_dev, optimize);
	}
	else
	{
		var_t* coeff = h_bh;
		tools::calc_lin_comb_s(f.yout, f.y, h_k, coeff, 6, f.n_var);
	}
}

void int_rungekutta5::calc_error(uint32_t n)
{
	if (PROC_UNIT_GPU == comp_dev.proc_unit)
	{
        gpu_calc_rk5_error(err, k[5], k[6], n, comp_dev.id_dev, optimize);
	}
	else
	{
		for (uint32_t i = 0; i < n; i++)
		{
			h_err[i] = fabs(k[5][i] - k[6][i]);
		}
	}
}

var_t int_rungekutta5::step()
{
	static std::string err_msg1 = "The integrator could not provide the approximation of the solution with the specified tolerance.";

	static const uint16_t n_a = sizeof(int_rungekutta5::a) / sizeof(var_t);
	static const uint16_t n_bh = sizeof(int_rungekutta5::bh) / sizeof(var_t);
	static bool first_call = true;
	static uint32_t n_var = 0;

    if (n_var != f.n_var)
	{
		optimize = true;
		n_var = f.n_var;
	}
	else
	{
		optimize = false;
	}

	uint16_t stage = 0;
	t = f.t;
	//f.calc_dy(stage, t, f.y, k[0]); // -> k1

	// The final function evaluation at the nth step is the same as the first at the (n+1)th step,
	// thus the effective number of function evaluations per step is 6.
	if (!adaptive)
	{
		// Calculate initial differentials and store them into k1
		f.calc_dy(stage, t, f.y, k[0]); // -> k1
	}
	else
	{
		if (first_call)
		{
			first_call = false;
			// Calculate initial differentials and store them into k1
			f.calc_dy(stage, t, f.y, k[0]); // -> k1
		}
		else
		{
            if (PROC_UNIT_GPU == comp_dev.proc_unit)
            {
                CUDA_SAFE_CALL(hipMemcpy(k[0], k[6], f.n_var*sizeof(var_t), hipMemcpyDeviceToDevice));
            }
            else
            {
    			memcpy(k[0], k[6], f.n_var*sizeof(var_t));
            }
		}
	}

	var_t max_err = 0.0;
	uint16_t iter = 0;
	do
	{
		dt_did = dt_try;
		// Compute in advance the dt_try * coefficients to save n_var multiplication per stage
		for (uint16_t i = 0; i < n_a; i++)
		{
			h_a[i] = dt_try * a[i];
		}
		for (uint16_t i = 0; i < n_bh; i++)
		{
			h_bh[i] = dt_try * bh[i];
		}
	    if (PROC_UNIT_GPU == comp_dev.proc_unit)
	    {
			copy_vector_to_device(d_a,  h_a,  sizeof(h_a) );
			copy_vector_to_device(d_bh, h_bh, sizeof(h_bh));
	    }

		for (stage = 1; stage < 6; stage++)
		{
			t = f.t + c[stage] * dt_try;
			calc_ytemp(stage);
			f.calc_dy(stage, t, ytemp, k[stage]); // -> k2, k3, k4, k5, k6
		}
		// We have stage (6) number of k vectors, approximate the solution in f.yout using the bh coeff:
		calc_y_np1();   // -> f.yout = y = ynp1 = yn + 17/192*k1 + ... + 1/20*k6

		if (adaptive)
		{
			// Here stage = 6
			t = f.t + c[stage] * dt_try;
			f.calc_dy(stage, t, f.yout, k[stage]); // -> k7
			calc_error(f.n_var);
			max_err = get_max_error(f.n_var);
			max_err *= dt_try * lambda;
			calc_dt_try(max_err);
		}
		iter++;
	} while (adaptive && max_iter > iter && dt_min < dt_try && max_err > tolerance);

	if (max_iter <= iter)
	{
		throw std::string(err_msg1 + " The number of iteration exceeded the limit.");
	}
	if (dt_min >= dt_try)
	{
		throw std::string(err_msg1 + " The stepsize is smaller than the limit.");
	}

	t = f.tout = f.t + dt_did;
	f.swap();

	update_counters(iter);

	return dt_did;
}
