#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>

#include "rtbp3D.h"

#include "redutil2.h"
#include "constants.h"

using namespace std;
using namespace redutil2;


rtbp3D::rtbp3D(uint16_t n_ppo, size_t omd_size, comp_dev_t comp_dev) :
	ode(3, 1, 9, n_ppo, omd_size, comp_dev)
{
	name = "Regularized 3D two-body problem";

	initialize();
}

rtbp3D::~rtbp3D()
{ }

void rtbp3D::initialize()
{
    h_md = (tbp_t::metadata_t*)h_omd;
    d_md = (tbp_t::metadata_t*)d_omd;
      md = (tbp_t::metadata_t*)omd;

	h       = 0.0;            // energy
	h_y[8]  = 0.0;            // s_0: fictitious time (4 position, 4 velocity, 1 time)
}

void rtbp3D::trans_to_descartes_var(var_t& x, var_t& y, var_t& z, var_t& vx, var_t& vy, var_t& vz)
{
	//var_t r = SQR(h_y[0]) + SQR(h_y[1]) + SQR(h_y[2]) + SQR(h_y[3]);							// r = u1^2 + u2^2 + u3^2 + u4^2

//	x  = SQR(h_y[0]) - SQR(h_y[1]) - SQR(h_y[2]) + SQR(h_y[3]);									// x = u1^2 - u2^2 - u3^2 + u4^2
//	y  = 2.0 * (h_y[0] * h_y[1] - h_y[2] * h_y[3]);												// y = 2*(u1*u2 - u3*u4)
//	z  = 2.0 * (h_y[0] * h_y[2] - h_y[1] * h_y[3]);												// z = 2*(u1*u3 - u2*u4)
//	vx = (2.0/r) * (h_y[0] * h_y[4] - h_y[1] * h_y[5] - h_y[2] * h_y[6] + h_y[3] * h_y[7]);		// vx = 2/r * (u1*vu1 - u2*vu2 - u3*vu3 + u4*vu4)
//	vy = (2.0/r) * (h_y[1] * h_y[4] + h_y[0] * h_y[5] - h_y[3] * h_y[6] - h_y[2] * h_y[7]);		// vy = 2/r * (u2*vu1 - u1*vu2 - u4*vu3 + u3*vu4)
//	vz = (2.0/r) * (h_y[2] * h_y[4] + h_y[3] * h_y[5] + h_y[0] * h_y[6] + h_y[1] * h_y[7]);		// vz = 2/r * (u3*vu1 - u4*vu2 - u1*vu3 + u2*vu4)
}

static void trans_to_descartes(const var4_t& u, const var4_t& u_prime, var3_t& r, var3_t& v)
{
	var_t d = SQR(u.x) + SQR(u.y) + SQR(u.z) + SQR(u.w);							// r = u1^2 + u2^2 + u3^2 + u4^2

	r.x  = SQR(u.x) - SQR(u.y) - SQR(u.z) + SQR(u.w);									// x = u1^2 - u2^2 - u3^2 + u4^2
//TODO
	//y  = 2.0 * (h_y[0] * h_y[1] - h_y[2] * h_y[3]);												// y = 2*(u1*u2 - u3*u4)
	//z  = 2.0 * (h_y[0] * h_y[2] - h_y[1] * h_y[3]);												// z = 2*(u1*u3 - u2*u4)
	//vx = (2.0/r) * (h_y[0] * h_y[4] - h_y[1] * h_y[5] - h_y[2] * h_y[6] + h_y[3] * h_y[7]);		// vx = 2/r * (u1*vu1 - u2*vu2 - u3*vu3 + u4*vu4)
	//vy = (2.0/r) * (h_y[1] * h_y[4] + h_y[0] * h_y[5] - h_y[3] * h_y[6] - h_y[2] * h_y[7]);		// vy = 2/r * (u2*vu1 - u1*vu2 - u4*vu3 + u3*vu4)
	//vz = (2.0/r) * (h_y[2] * h_y[4] + h_y[3] * h_y[5] + h_y[0] * h_y[6] + h_y[1] * h_y[7]);		// vz = 2/r * (u3*vu1 - u4*vu2 - u1*vu3 + u2*vu4)
}

//void rtbp3D::trans_to_regular_var(var_t& x, var_t& y, var_t& z, var_t& vx, var_t& vy, var_t& vz)
//{
//	var_t r = SQR(h_y[0]) + SQR(h_y[1]) + SQR(h_y[2]) + SQR(h_y[3]);							// r = u1^2 + u2^2 + u3^2 + u4^2
//
//	x  = SQR(h_y[0]) - SQR(h_y[1]) - SQR(h_y[2]) + SQR(h_y[3]);									// x = u1^2 - u2^2 - u3^2 + u4^2
//	y  = 2.0 * (h_y[0] * h_y[1] - h_y[2] * h_y[3]);												// y = 2*(u1*u2 - u3*u4)
//	z  = 2.0 * (h_y[0] * h_y[2] - h_y[1] * h_y[3]);												// z = 2*(u1*u3 - u2*u4)
//	vx = (2.0/r) * (h_y[0] * h_y[4] - h_y[1] * h_y[5] - h_y[2] * h_y[6] + h_y[3] * h_y[7]);		// vx = 2/r * (u1*vu1 - u2*vu2 - u3*vu3 + u4*vu4)
//	vy = (2.0/r) * (h_y[1] * h_y[4] + h_y[0] * h_y[5] - h_y[3] * h_y[6] - h_y[2] * h_y[7]);		// vy = 2/r * (u2*vu1 - u1*vu2 - u4*vu3 + u3*vu4)
//	vz = (2.0/r) * (h_y[2] * h_y[4] + h_y[3] * h_y[5] + h_y[0] * h_y[6] + h_y[1] * h_y[7]);		// vz = 2/r * (u3*vu1 - u4*vu2 - u1*vu3 + u2*vu4)
//}

void rtbp3D::calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* acc, var_t* jrk)
{
	throw string("The rtbp3D::calc_dy is not implemented.");
}

void rtbp3D::calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	if (PROC_UNIT_CPU == comp_dev.proc_unit)
	{
		cpu_calc_dy(stage, curr_t, y_temp, dy);
	}
	else
	{
		gpu_calc_dy(stage, curr_t, y_temp, dy);
	}
}

void rtbp3D::calc_integral()
{
	static bool first_call = true;

	const tbp_t::param_t* p = (tbp_t::param_t*)h_par;

	var_t r  = SQR(h_y[0]) + SQR(h_y[1]) + SQR(h_y[2]) + SQR(h_y[3]);
	var_t vx = (2.0/r) * (h_y[0] * h_y[4] - h_y[1] * h_y[5] - h_y[2] * h_y[6] + h_y[3] * h_y[7]);		// vx = 2/r * (u1*vu1 - u2*vu2 - u3*vu3 + u4*vu4)
	var_t vy = (2.0/r) * (h_y[1] * h_y[4] + h_y[0] * h_y[5] - h_y[3] * h_y[6] - h_y[2] * h_y[7]);		// vy = 2/r * (u2*vu1 - u1*vu2 - u4*vu3 + u3*vu4)
	var_t vz = (2.0/r) * (h_y[2] * h_y[4] + h_y[3] * h_y[5] + h_y[0] * h_y[6] + h_y[1] * h_y[7]);		// vz = 2/r * (u3*vu1 - u4*vu2 - u1*vu3 + u2*vu4)
	var_t v2 = SQR(vx) + SQR(vy) + SQR(vz);
	h = 0.5 * v2 - p[0].mu / r;

	if (first_call)
	{
		integral.h0 = integral.h;
		first_call = false;
	}
}

void rtbp3D::cpu_calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	dy[0] = y_temp[4];																// dy1 / ds = y5
	dy[1] = y_temp[5];																// dy2 / ds = y6
	dy[2] = y_temp[6];																// dy3 / ds = y7
	dy[3] = y_temp[7];																// dy4 / ds = y8

	dy[4] = (h / 2.0) * y_temp[0];													// dy5 / ds = h/2 * y1
	dy[5] = (h / 2.0) * y_temp[1];													// dy6 / ds = h/2 * y2
	dy[6] = (h / 2.0) * y_temp[2];													// dy7 / ds = h/2 * y3
	dy[7] = (h / 2.0) * y_temp[3];													// dy8 / ds = h/2 * y4

	dy[8] = SQR(y_temp[0]) + SQR(y_temp[1])  + SQR(y_temp[2]) + SQR(y_temp[3]);     // dy9 / ds = y1^2 + y2^2 + y3^2 + y4^2
}

void rtbp3D::gpu_calc_dy(uint16_t stage, var_t curr_t, const var_t* y_temp, var_t* dy)
{
	throw string("The gpu_calc_dy() is not implemented.");
}

void rtbp3D::load(string& path)
{
	ifstream input;

	cout << "Loading " << path << " ";

	data_rep_t repres = (file::get_extension(path) == "txt" ? DATA_REPRESENTATION_ASCII : DATA_REPRESENTATION_BINARY);
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		input.open(path.c_str());
		if (input) 
		{
			load_ascii(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		input.open(path.c_str(), ios::binary);
		if (input) 
		{
			load_binary(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	}
	input.close();

	cout << " done" << endl;
}

void rtbp3D::load_ascii(ifstream& input)
{
	tbp_t::param_t* p = (tbp_t::param_t*)h_par;

	var_t _t;
	for (uint32_t i = 0; i < n_obj; i++)
	{
		load_ascii_record(input, &_t, &h_md[i], &p[i], &h_y[i], &h_y[i+4]);
	}
}

void rtbp3D::load_ascii_record(ifstream& input, var_t* t, tbp_t::metadata_t *md, tbp_t::param_t* p, var_t* r, var_t* v)
{
	string name;

	// epoch
	input >> *t;
	// id
	input >> md->id;
	// mu = k^2*(m1 + m2)
	input >> p->mu;

	// position
	var4_t* _r = (var4_t*)r;
	input >> _r->x >> _r->y >> _r->z >> _r->w;
	// velocity
	var4_t* _v = (var4_t*)v;
	input >> _v->x >> _v->y >> _v->z >> _v->w;

}

void rtbp3D::load_binary(ifstream& input)
{
	throw string("The load_binary() is not implemented.");
}

void rtbp3D::print_solution(std::string& path_si, std::string& path_sd, data_rep_t repres)
{
	ofstream sout;

	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		sout.open(path_si.c_str(), ios::out | ios::app);
		break;
	case DATA_REPRESENTATION_BINARY:
		sout.open(path_si.c_str(), ios::out | ios::app | ios::binary);
		break;
	default:
		throw string("Parameter 'repres' is out of range.");
	}

	if (sout)
	{
		switch (repres)
		{
		case DATA_REPRESENTATION_ASCII:
			print_solution_ascii(sout);
			break;
		case DATA_REPRESENTATION_BINARY:
			print_solution_binary(sout);
			break;
		default:
			throw string("Parameter 'repres' is out of range.");
		}
	}
	else
	{
		throw string("Cannot open " + path_si + ".");
	}
	sout.close();
}

void rtbp3D::print_dump(std::string& path_si, std::string& path_sd)
{
    throw string("The rtbp3D::print_dump function is not implemented.");
}

void rtbp3D::print_solution_ascii(ofstream& sout) //TODO: implement correctly
{	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	var_t x = 0.0;
	var_t y = 0.0;
	var_t z = 0.0;
	var_t vx = 0.0;
	var_t vy = 0.0;
	var_t vz = 0.0;
	trans_to_descartes_var(x, y, z, vx, vy, vz);

	for (uint32_t i = 0; i < n_obj; i++)
    {
		sout << setw(VAR_T_W) << t << SEP                       /* time of the record [day] (double)           */
		// Print the metadata for each object
        << setw(INT_T_W) << h_md[i].id << SEP;

		// Print the parameters for each object
		for (uint16_t j = 0; j < n_ppo; j++)
		{
			uint32_t param_idx = i * n_ppo + j;
			sout << setw(VAR_T_W) << h_par[param_idx] << SEP;
		}
		// Print the regularized variables for each object
		for (uint16_t j = 0; j < n_vpo; j++)
		{
			uint32_t var_idx = i * n_vpo + j;
			sout << setw(VAR_T_W) << h_y[var_idx] << SEP;
		}
		// Print the descartes non-regularized variables for each object
		sout << setw(VAR_T_W) << x << SEP << y << SEP << z << SEP
			 << setw(VAR_T_W) << vx << SEP << vy << SEP << vz << endl;
	}
	sout.flush();
}

void rtbp3D::print_solution_binary(ofstream& sout)
{
	throw string("The print_solution_binary() is not implemented.");
}

void rtbp3D::print_integral(string& path)
{
	ofstream sout;

	sout.open(path.c_str(), ios::out | ios::app);
	if (sout)
	{
		sout.precision(16);
		sout.setf(ios::right);
		sout.setf(ios::scientific);

		sout << setw(VAR_T_W) << t << SEP                       /* fictitious time of the record (double)           */
			 << setw(VAR_T_W) << h_y[8] << SEP                  /* real time of the record [day] double             */
			 << h << endl;                                      /* energy of the system                             */
	}
	else
	{
		throw string("Cannot open " + path + ".");
	}
	sout.close();
}
