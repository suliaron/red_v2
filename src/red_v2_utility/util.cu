#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <ostream>
#include <sstream>
#include <string>

#ifdef _WIN32
#include <chrono>
#include <Windows.h>
#else
#include <sys/time.h>
#include <ctime>
#endif

#include "hip/hip_runtime.h"
#include ""

#include "util.h"
#include "file_util.h"
#include "type.h"
#include "macro.h"

using namespace std;

namespace red_kernel
{
//! Print pointer and dereferennce it
__global__
void print_ptr(const var_t* a)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (0 == tid)
	{
		printf("%p\t%25.16le\n", a, *a);
	}
}

__global__
void print_ptr(const var_t* const *a)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (0 == tid)
	{
		printf("%p\t%p\n", a, *a);
	}
}
	
//! Calculate the special linear combination of two vectors, a[i] = b[i] + f*c[i]
__global__
void calc_lin_comb_s(var_t* a, const var_t* b, var_t f, const var_t* c, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		a[tid] = b[tid] + f * c[tid];
		tid += stride;
	}
}

//! Calculate the special case of linear combination of vectors, a[i] = b[i] + sum (coeff[j] * c[j][i])
__global__
void calc_lin_comb_s(var_t* a, const var_t* b, const var_t* const *c, const var_t* coeff, uint16_t n_vct, uint32_t n_var)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < n_var)
	{
		var_t d = 0.0;
		for (uint16_t j = 0; j < n_vct; j++)
		{
			if (0.0 == coeff[j])
			{
				continue;
			}
			d += coeff[j] * c[j][tid];
		}
		a[tid] = b[tid] + d;
	}
}

//! Calculate the error for the Runge-Kutta 4 method: error = |k4 - k5|
__global__
void calc_rk4_error(var_t* a, const var_t* k4, const var_t* k5, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
        a[tid] = fabs(k4[tid] - k5[tid]);
		tid += stride;
	}
}

//! Calculate the error for the Runge-Kutta 5 method: error = |k5 - k6|
__global__
void calc_rk5_error(var_t* a, const var_t* k5, const var_t* k6, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
        a[tid] = fabs(k5[tid] - k6[tid]);
		tid += stride;
	}
}

//! Calculate the error for the Runge-Kutta 7 method: error = |k1 + k11 - k12 - k13|
__global__
void calc_rk7_error(var_t* a, const var_t* k1, const var_t* k11, const var_t* k12, const var_t* k13, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
        a[tid] = fabs(k1[tid] + k11[tid] - k12[tid] - k13[tid]);
		tid += stride;
	}
}
} /* red_kernel */

namespace redutil2
{
/*
* Returns the amount of microseconds elapsed since the UNIX epoch.
* Works on windows and linux.
*/
uint64_t GetTimeMs64()
{
#ifdef _WIN32
    FILETIME ft;
    LARGE_INTEGER li;

    /* Get the amount of 100 nano seconds intervals elapsed since January 1, 1601 (UTC)
    * and copy it to a LARGE_INTEGER structure.
    */
    GetSystemTimeAsFileTime(&ft);
    li.LowPart = ft.dwLowDateTime;
    li.HighPart = ft.dwHighDateTime;

    uint64_t ret = li.QuadPart;
    /* Convert from file time to UNIX epoch time. */
    ret -= 116444736000000000LL;
    ret /= 10;      /* From 100 nano seconds (10^-7) to 1 microsecond (10^-6) intervals */

    return ret;
#else
    // Linux
    struct timeval tv;

    gettimeofday(&tv, NULL);
    uint64_t ret = tv.tv_usec;
    /* Convert from micro seconds (10^-6) to milliseconds (10^-3) */
    //ret /= 1000;

    /* Adds the seconds (10^0) after converting them to microseconds (10^-6) */
    ret += (tv.tv_sec * 1000000);

    return ret;
#endif
}

    template <typename T>
std::string number_to_string( T number, uint32_t width, bool fill)
{
	std::ostringstream ss;

	if (fill)
	{
		if (0 < width)
		{
			ss << setw(width) << setfill('0') << number;
		}
		else
		{
			ss << setfill('0') << number;
		}
	}
	else
	{
		if (0 < width)
		{
			ss << setw(width) << number;
		}
		else
		{
			ss << number;
		}
	}

	return ss.str();
}

template std::string number_to_string<char>(                  char, uint32_t width, bool fill);
template std::string number_to_string<unsigned char>(unsigned char, uint32_t width, bool fill);
template std::string number_to_string<int>(                    int, uint32_t width, bool fill);
template std::string number_to_string<uint32_t>(  uint32_t, uint32_t width, bool fill);
template std::string number_to_string<long>(                  long, uint32_t width, bool fill);
template std::string number_to_string<unsigned long>(unsigned long, uint32_t width, bool fill);

template <typename T>
std::string number_to_string( T number )
{
	std::ostringstream ss;
	ss << number;
	return ss.str();
}

template std::string number_to_string<char>(char);
template std::string number_to_string<unsigned char>(unsigned char);
template std::string number_to_string<int>(int);
template std::string number_to_string<uint32_t>(uint32_t);
template std::string number_to_string<long>(long);
template std::string number_to_string<unsigned long>(unsigned long);
template std::string number_to_string<float>(float);
template std::string number_to_string<double>(double);

__host__ __device__
	var4_t rotate_2D_vector(var_t theta, const var4_t& r)
{
	var_t ct = cos(theta);
	var_t st = sin(theta);

	var4_t result = {ct * r.x - st * r.y, st * r.x + ct * r.y, 0.0, 0.0};
	return result;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        {   -1, -1 }
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
    return nGpuArchCoresPerSM[7].Cores;
}
// end of GPU Architecture definitions

string get_device_name(int id_dev)
{
	hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, id_dev));

	string result(deviceProp.name);
	return result;
}

// TODO: implement
int get_id_fastest_cuda_device()
{
	return 0;
}

int get_n_cuda_device()
{
	int n_device = 0;
	CUDA_SAFE_CALL(hipGetDeviceCount(&n_device));
	return n_device;
}

void device_query(ostream& sout, int id_dev)
{
    int deviceCount = 0;
    CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount));

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
		throw string("There are no available device(s) that support CUDA.");
    }

    int dev, driverVersion = 0, runtimeVersion = 0;

    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, id_dev));

    sout << "The code runs on a " << deviceProp.name << " device:" << endl;

    // Console log
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
	sout << "  CUDA Driver Version / Runtime Version          " << driverVersion/1000 << "." << (driverVersion%100)/10 << " / " << runtimeVersion/1000 << "." << (runtimeVersion%100)/10 << endl;
	sout << "  CUDA Capability Major/Minor version number:    " << deviceProp.major << "." << deviceProp.minor << endl;
	sout << "  Total amount of global memory:                 " << deviceProp.totalGlobalMem/1048576.0f << " MBytes" << endl;
	sout << "  " << deviceProp.multiProcessorCount <<  " Multiprocessors, " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) << " CUDA Cores/MP:     " << _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount << " CUDA Cores" << endl;
	sout << "  GPU Clock rate:                                " << deviceProp.clockRate * 1e-3f << ".0 MHz" << endl;
	sout << "  Total amount of constant memory:               " << deviceProp.totalConstMem << " bytes" << endl;
	sout << "  Total amount of shared memory per block:       " << deviceProp.sharedMemPerBlock << " bytes" << endl;
	sout << "  Total number of registers available per block: " << deviceProp.regsPerBlock << endl;
	sout << "  Warp size:                                     " << deviceProp.warpSize << endl;
	sout << "  Maximum number of threads per multiprocessor:  " << deviceProp.maxThreadsPerMultiProcessor << endl;
	sout << "  Maximum number of threads per block:           " << deviceProp.maxThreadsPerBlock << endl;
	sout << "  Max dimension size of a thread block (x,y,z): (" << deviceProp.maxThreadsDim[0] << "," << deviceProp.maxThreadsDim[1] << "," << deviceProp.maxThreadsDim[2] << ")" << endl;
	sout << "  Max dimension size of a grid size    (x,y,z): (" <<deviceProp.maxGridSize[0] << "," << deviceProp.maxGridSize[1] << "," << deviceProp.maxGridSize[2] << ")" << endl << endl;

    std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[16];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#else
    sprintf(cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Device count
    sProfileString += ", NumDevs = ";
#ifdef WIN32
    sprintf_s(cTemp, 10, "%d", deviceCount);
#else
    sprintf(cTemp, "%d", deviceCount);
#endif
    sProfileString += cTemp;

    // Print Out all device Names
    for (dev = 0; dev < deviceCount; ++dev)
    {
#ifdef _WIN32
        sprintf_s(cTemp, 13, ", Device%d = ", dev);
#else
        sprintf(cTemp, ", Device%d = ", dev);
#endif
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        sProfileString += cTemp;
        sProfileString += deviceProp.name;
    }
    sProfileString += "\n";

	sout << sProfileString;
}

void set_kernel_launch_param(uint32_t n_data, uint16_t n_tpb, dim3& grid, dim3& block)
{
	uint32_t n_thread = min((uint32_t)n_tpb, n_data);
	uint32_t n_block = (n_data + n_thread - 1)/n_thread;

	grid.x	= n_block;
	block.x = n_thread;
}

void device_query(ostream& sout, int id_dev, bool print_to_screen)
{
	device_query(sout, id_dev);
	if (print_to_screen)
	{
		device_query(cout, id_dev);
	}
}

void allocate_host_vector(void **ptr, size_t size, const char *file, int line)
{
	*ptr = (void *)malloc(size);
	if (NULL == ptr)
	{
		throw string("Out of memory.");
	}

	// Clear memory 
	memset(*ptr, 0, size);
}

void allocate_device_vector(void **ptr, size_t size, const char *file, int line)
{
	// Allocate memory
	CUDA_SAFE_CALL(hipMalloc(ptr, size));
	// Clear memory 
	CUDA_SAFE_CALL(hipMemset(*ptr, 0, size));
}

void allocate_vector(void **ptr, size_t size, bool cpu, const char *file, int line)
{
	if (cpu)
	{
		allocate_host_vector(ptr, size, file, line);
	}
	else
	{
		allocate_device_vector(ptr, size, file, line);
	}
}

void free_host_vector(void **ptr, const char *file, int line)
{
	if (NULL != *ptr)
	{
		delete[] *ptr;
		*ptr = NULL;
	}
}

void free_device_vector(void **ptr, const char *file, int line)
{
	if (NULL != *ptr)
	{
		CUDA_SAFE_CALL(hipFree(*ptr));
		*ptr = NULL;
	}
}

//void free_vector(void **ptr, bool cpu, const char *file, int line)
//{
//	if (cpu)
//	{
//		free_host_vector(ptr, file, line);
//	}
//	else
//	{
//		free_device_vector(ptr, file, line);
//	}
//}

void allocate_host_storage(pp_disk_t::sim_data_t *sd, int n)
{
	sd->h_y.resize(2);
	sd->h_yout.resize(2);

	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_HOST_VECTOR((void **)&(sd->h_y[i]),    n*sizeof(var4_t));
		ALLOCATE_HOST_VECTOR((void **)&(sd->h_yout[i]), n*sizeof(var4_t));
	}
	ALLOCATE_HOST_VECTOR((void **)&(sd->h_p),           n*sizeof(pp_disk_t::param_t));
	ALLOCATE_HOST_VECTOR((void **)&(sd->h_body_md),     n*sizeof(pp_disk_t::body_metadata_t));
	ALLOCATE_HOST_VECTOR((void **)&(sd->h_epoch),       n*sizeof(var_t));

	ALLOCATE_HOST_VECTOR((void **)&(sd->h_oe),          n*sizeof(orbelem_t));
}

void allocate_device_storage(pp_disk_t::sim_data_t *sd, int n)
{
	sd->d_y.resize(2);
	sd->d_yout.resize(2);

	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_y[i]),	  n*sizeof(var4_t));
		ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_yout[i]), n*sizeof(var4_t));
	}
	ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_p),			  n*sizeof(pp_disk_t::param_t));
	ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_body_md),	  n*sizeof(pp_disk_t::body_metadata_t));
	ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_epoch),		  n*sizeof(var_t));

    ALLOCATE_DEVICE_VECTOR((void **)&(sd->d_oe),          n*sizeof(orbelem_t));
}

void deallocate_host_storage(pp_disk_t::sim_data_t *sd)
{
	for (int i = 0; i < 2; i++)
	{
		FREE_HOST_VECTOR((void **)&(sd->h_y[i]));
		FREE_HOST_VECTOR((void **)&(sd->h_yout[i]));
	}
	FREE_HOST_VECTOR((void **)&(sd->h_p));
	FREE_HOST_VECTOR((void **)&(sd->h_body_md));
	FREE_HOST_VECTOR((void **)&(sd->h_epoch));

	FREE_HOST_VECTOR((void **)&(sd->h_oe));
}

void deallocate_device_storage(pp_disk_t::sim_data_t *sd)
{
	for (int i = 0; i < 2; i++)
	{
		FREE_DEVICE_VECTOR((void **)&(sd->d_y[i]));
		FREE_DEVICE_VECTOR((void **)&(sd->d_yout[i]));
	}
	FREE_DEVICE_VECTOR((void **)&(sd->d_p));
	FREE_DEVICE_VECTOR((void **)&(sd->d_body_md));
	FREE_DEVICE_VECTOR((void **)&(sd->d_epoch));

    FREE_DEVICE_VECTOR((void **)&(sd->d_oe));
}


void copy_vector_to_device(void* dst, const void *src, size_t count)
{
	CUDA_SAFE_CALL(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
}

void copy_vector_to_host(void* dst, const void *src, size_t count)
{
	CUDA_SAFE_CALL(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
}

void copy_vector_d2d(void* dst, const void *src, size_t count)
{
	CUDA_SAFE_CALL(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice));
}

void copy_constant_to_device(const void* dst, const void *src, size_t count)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dst), src, count, 0, hipMemcpyHostToDevice));
}


void set_device(int id_of_target_dev, ostream& sout)
{
	int n_device = get_n_cuda_device();
	if (0 == n_device)
	{
		throw string("There are no available device(s) that support CUDA.");
	}

	if (n_device > id_of_target_dev && 0 <= id_of_target_dev)
	{
		// Set the desired id of the device
		CUDA_SAFE_CALL(hipSetDevice(id_of_target_dev));
	}
	else
	{
		throw string("The device with the requested id does not exist.");
	}
}

void print_array(string path, string comment, uint32_t n, var_t *data, mem_loc_t mem_loc)
{
	var_t* h_data = NULL;

	ostream *out = NULL;
	if (0 < path.length())
	{
		out = new ofstream(path.c_str(), ios::app);
	}
	else
	{
		out = &cout;
	}

	out->setf(ios::right);
	out->setf(ios::scientific);

	if (MEM_LOC_DEVICE == mem_loc)
	{
		h_data = new var_t[n];
		copy_vector_to_host(h_data, data, n * sizeof(var_t));
	}
	else
	{
		h_data = data;
	}
    if (!comment.empty()) *out << comment << endl;
    for (uint32_t i = 0; i < n; i++)
	{
		*out << setw(5) << i << setprecision(16) << setw(25) << h_data[i] << endl;
	}

	if (MEM_LOC_DEVICE == mem_loc)
	{
		delete[] h_data;
	}
	if (0 < path.length())
	{
		out->flush();
		delete out;
	}
}

void create_aliases(comp_dev_t comp_dev, pp_disk_t::sim_data_t *sd)
{
	switch (comp_dev.proc_unit)
	{
	case PROC_UNIT_CPU:
		for (int i = 0; i < 2; i++)
		{
			sd->y[i]    = sd->h_y[i];
			sd->yout[i] = sd->h_yout[i];
		}
		sd->p       = sd->h_p;
		sd->body_md = sd->h_body_md;
		sd->epoch   = sd->h_epoch;
        sd->oe      = sd->h_oe;
		break;
	case PROC_UNIT_GPU:
		for (int i = 0; i < 2; i++)
		{
			sd->y[i]    = sd->d_y[i];
			sd->yout[i] = sd->d_yout[i];
		}
		sd->p       = sd->d_p;
		sd->body_md = sd->d_body_md;
		sd->epoch   = sd->d_epoch;
        sd->oe      = sd->d_oe;
		break;
	default:
		throw string("Parameter 'proc_unit' is out of range.");
	}
}

// Date of creation: 2016.11.22.
// Last edited: 
// Status:
void gpu_calc_lin_comb_s(var_t* a, const var_t* b, const var_t* c, var_t f, uint32_t n_var, int id_dev, bool optimize)
{
	static uint16_t n_tpb = 256;
	static bool first_call = true;

	dim3 grid;
	dim3 block;

	if (optimize || first_call)
	{
		first_call = false;

		hipDeviceProp_t prop;
		hipEvent_t start, stop;

        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, id_dev));
		CUDA_SAFE_CALL(hipEventCreate(&start));
		CUDA_SAFE_CALL(hipEventCreate(&stop));

		float min_GPU_DT = 1.0e8;
		uint16_t d_nt = prop.warpSize / 2;
		for (uint16_t nt = d_nt; nt <= prop.maxThreadsPerBlock / 2; nt += d_nt)
		{
			set_kernel_launch_param(n_var, nt, grid, block);

			CUDA_SAFE_CALL(hipEventRecord(start));
			red_kernel::calc_lin_comb_s<<<grid, block>>>(a, b, f, c, n_var);
       		CUDA_CHECK_ERROR();
			CUDA_SAFE_CALL(hipEventRecord(stop));
			CUDA_SAFE_CALL(hipEventSynchronize(stop));

			float GPU_DT = 0.0f;
			CUDA_SAFE_CALL(hipEventElapsedTime(&GPU_DT, start, stop));
			if (GPU_DT < min_GPU_DT)
			{
				min_GPU_DT = GPU_DT;
				n_tpb = nt;
			}
			//printf("%4u %10.6f [ms]\n", nt, GPU_DT);
		}
		//printf("\n%4u %10.6f [ms]\n", n_tpb, min_GPU_DT);
	}
	else
	{
		set_kernel_launch_param(n_var, n_tpb, grid, block);
		red_kernel::calc_lin_comb_s<<<grid, block>>>(a, b, f, c, n_var);
   		CUDA_CHECK_ERROR();
	}
}

void gpu_calc_lin_comb_s(var_t* a, const var_t* b, const var_t* const *c, const var_t* coeff, uint16_t n_vct, uint32_t n_var, int id_dev, bool optimize)
{
	static uint16_t n_tpb = 256;
	static bool first_call = true;

	dim3 grid;
	dim3 block;

	if (optimize || first_call)
	{
		first_call = false;

		hipDeviceProp_t prop;
		hipEvent_t start, stop;

        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, id_dev));
		CUDA_SAFE_CALL(hipEventCreate(&start));
		CUDA_SAFE_CALL(hipEventCreate(&stop));

		float min_GPU_DT = 1.0e8;
		uint16_t d_nt = prop.warpSize / 2;
		for (uint16_t nt = d_nt; nt <= prop.maxThreadsPerBlock / 2; nt += d_nt)
		{
			set_kernel_launch_param(n_var, nt, grid, block);

            CUDA_SAFE_CALL(hipEventRecord(start));
			red_kernel::calc_lin_comb_s<<<grid, block>>>(a, b, c, coeff, n_vct, n_var);
    		CUDA_CHECK_ERROR();
			CUDA_SAFE_CALL(hipEventRecord(stop));
			CUDA_SAFE_CALL(hipEventSynchronize(stop));

			float GPU_DT = 0.0f;
			CUDA_SAFE_CALL(hipEventElapsedTime(&GPU_DT, start, stop));
			if (GPU_DT < min_GPU_DT)
			{
				min_GPU_DT = GPU_DT;
				n_tpb = nt;
			}
			//printf("%4u %10.6f [ms]\n", nt, GPU_DT);
		}
		//printf("\n%4u %10.6f [ms]\n", n_tpb, min_GPU_DT);
	}
	else
	{
		set_kernel_launch_param(n_var, n_tpb, grid, block);
		red_kernel::calc_lin_comb_s<<<grid, block>>>(a, b, c, coeff, n_vct, n_var);
   		CUDA_CHECK_ERROR();
	}
}

// Calculate the error for the Runge-Kutta 4 method: a = |k4 - k5|
void gpu_calc_rk4_error(var_t* a, const var_t* k4, const var_t* k5, uint32_t n_var, int id_dev, bool optimize)
{
	static uint16_t n_tpb = 256;
	static bool first_call = true;

	dim3 grid;
	dim3 block;

	if (optimize || first_call)
	{
		first_call = false;

		hipDeviceProp_t prop;
		hipEvent_t start, stop;

        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, id_dev));
		CUDA_SAFE_CALL(hipEventCreate(&start));
		CUDA_SAFE_CALL(hipEventCreate(&stop));

		float min_GPU_DT = 1.0e8;
		uint16_t d_nt = prop.warpSize / 2;
		for (uint16_t nt = d_nt; nt <= prop.maxThreadsPerBlock / 2; nt += d_nt)
		{
			set_kernel_launch_param(n_var, nt, grid, block);

			CUDA_SAFE_CALL(hipEventRecord(start));
            red_kernel::calc_rk4_error<<<grid, block>>>(a, k4, k5, n_var);
       		CUDA_CHECK_ERROR();
			CUDA_SAFE_CALL(hipEventRecord(stop));
			CUDA_SAFE_CALL(hipEventSynchronize(stop));

			float GPU_DT = 0.0f;
			CUDA_SAFE_CALL(hipEventElapsedTime(&GPU_DT, start, stop));
			if (GPU_DT < min_GPU_DT)
			{
				min_GPU_DT = GPU_DT;
				n_tpb = nt;
			}
			//printf("%4u %10.6f [ms]\n", nt, GPU_DT);
		}
		//printf("\n%4u %10.6f [ms]\n", n_tpb, min_GPU_DT);
	}
	else
	{
		set_kernel_launch_param(n_var, n_tpb, grid, block);
        red_kernel::calc_rk4_error<<<grid, block>>>(a, k4, k5, n_var);
   		CUDA_CHECK_ERROR();
	}
}

// Calculate the error for the Runge-Kutta 5 method: a = |k5 - k6|
void gpu_calc_rk5_error(var_t* a, const var_t* k5, const var_t* k6, uint32_t n_var, int id_dev, bool optimize)
{
	static uint16_t n_tpb = 256;
	static bool first_call = true;

	dim3 grid;
	dim3 block;

	if (optimize || first_call)
	{
		first_call = false;

		hipDeviceProp_t prop;
		hipEvent_t start, stop;

        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, id_dev));
		CUDA_SAFE_CALL(hipEventCreate(&start));
		CUDA_SAFE_CALL(hipEventCreate(&stop));

		float min_GPU_DT = 1.0e8;
		uint16_t d_nt = prop.warpSize / 2;
		for (uint16_t nt = d_nt; nt <= prop.maxThreadsPerBlock / 2; nt += d_nt)
		{
			set_kernel_launch_param(n_var, nt, grid, block);

			CUDA_SAFE_CALL(hipEventRecord(start));
            red_kernel::calc_rk5_error<<<grid, block>>>(a, k5, k6, n_var);
       		CUDA_CHECK_ERROR();
			CUDA_SAFE_CALL(hipEventRecord(stop));
			CUDA_SAFE_CALL(hipEventSynchronize(stop));

			float GPU_DT = 0.0f;
			CUDA_SAFE_CALL(hipEventElapsedTime(&GPU_DT, start, stop));
			if (GPU_DT < min_GPU_DT)
			{
				min_GPU_DT = GPU_DT;
				n_tpb = nt;
			}
			//printf("%4u %10.6f [ms]\n", nt, GPU_DT);
		}
		//printf("\n%4u %10.6f [ms]\n", n_tpb, min_GPU_DT);
	}
	else
	{
		set_kernel_launch_param(n_var, n_tpb, grid, block);
        red_kernel::calc_rk5_error<<<grid, block>>>(a, k5, k6, n_var);
   		CUDA_CHECK_ERROR();
	}
}

// Calculate the error for the Runge-Kutta 7 method: a = |k1 + k11 - k12 - k13|
void gpu_calc_rk7_error(var_t* a, const var_t* k1, const var_t* k11, const var_t* k12, const var_t* k13, uint32_t n_var, int id_dev, bool optimize)
{
	static uint16_t n_tpb = 256;
	static bool first_call = true;

	dim3 grid;
	dim3 block;

	if (optimize || first_call)
	{
		first_call = false;

		hipDeviceProp_t prop;
		hipEvent_t start, stop;

        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, id_dev));
		CUDA_SAFE_CALL(hipEventCreate(&start));
		CUDA_SAFE_CALL(hipEventCreate(&stop));

		float min_GPU_DT = 1.0e8;
		uint16_t d_nt = prop.warpSize / 2;
		for (uint16_t nt = d_nt; nt <= prop.maxThreadsPerBlock / 2; nt += d_nt)
		{
			set_kernel_launch_param(n_var, nt, grid, block);

			CUDA_SAFE_CALL(hipEventRecord(start));
            red_kernel::calc_rk7_error<<<grid, block>>>(a, k1, k11, k12, k13, n_var);
       		CUDA_CHECK_ERROR();
			CUDA_SAFE_CALL(hipEventRecord(stop));
			CUDA_SAFE_CALL(hipEventSynchronize(stop));

			float GPU_DT = 0.0f;
			CUDA_SAFE_CALL(hipEventElapsedTime(&GPU_DT, start, stop));
			if (GPU_DT < min_GPU_DT)
			{
				min_GPU_DT = GPU_DT;
				n_tpb = nt;
			}
			//printf("%4u %10.6f [ms]\n", nt, GPU_DT);
		}
		//printf("\n%4u %10.6f [ms]\n", n_tpb, min_GPU_DT);
	}
	else
	{
		set_kernel_launch_param(n_var, n_tpb, grid, block);
        red_kernel::calc_rk7_error<<<grid, block>>>(a, k1, k11, k12, k13, n_var);
   		CUDA_CHECK_ERROR();
	}
}
} /* redutil2 */
